#include "hip/hip_runtime.h"

#include <string>
#include <math.h>

#include <stdio.h>

#define numBlocks 12
#define numThreads 32

struct RSA_KEY
{
	unsigned long p; // selected prime 1
	unsigned long q; // selected prime 2
	unsigned long n; // public - the modulus
	unsigned long e; // public - for encryption
	unsigned long d; // private - for decryption
};

// Function prototypes
RSA_KEY generate_RSA_key(unsigned long p, unsigned long q);
void print_RSA_key(RSA_KEY in_key);
void RSA_encode(
	char *input,
	size_t input_size,
	unsigned long long *output,
	size_t output_size,
	unsigned long e,
	unsigned long n);
void RSA_decode(
	unsigned long long *input,
	size_t input_size,
	char *output,
	size_t output_size,
	unsigned long d,
	unsigned long n);
int gcd(int a, int b);
int modulo(int a, int b, int n);
__device__ int is_prime(unsigned long input);

// RSA Cracking Kernel
__global__ void findPrime(unsigned long n, unsigned long roundedN)
{
	// Round the input modulus to nearest power of 2
	unsigned long rangeRounded = 2 << roundedN;
	
	// Sanity dictates that both primes should be < half the modulus
	unsigned long rangeTotal = rangeRounded / 2;

	// Determine min & max range for this thread
	unsigned long index = blockIdx.x * numThreads + threadIdx.x;
	unsigned long rangeLow = rangeTotal / (numBlocks * numThreads) * index;
	unsigned long rangeHigh = rangeTotal / (numBlocks * numThreads) * (index + 1) - 1;

	//printf("Thread %d reporting in N:%d | %d to %d\n", index, n, rangeLow, rangeHigh);

	// Loop through range and search for primes
	unsigned long output = 0;
	for (unsigned long myindex = rangeLow; myindex < rangeHigh; myindex++)
	{
		if (is_prime(myindex))
		{
			if (n % myindex == 0)
			{
				output = myindex;
				printf("prime: %d\n", myindex);
			}
		}
	}

	// Debug Print
	if (output != 0)
		printf("B:%d T:%d I:%d Range: %8d to %8d of %8d RESULT: %d\n", 
			blockIdx.x, threadIdx.x, index, rangeLow, rangeHigh, rangeTotal, output);
}

int main()
{
	// Message to encode
	char secret_message[] = "The quick brown fox jumped over the lazy dog.";
	printf("Message: %s\n\n",secret_message);

	// Generate public & private key
	printf("Generating key...\n");
	RSA_KEY my_key;
	unsigned long prime1 = 157;
	unsigned long prime2 = 199;
	my_key = generate_RSA_key(prime1, prime2);
	print_RSA_key(my_key);

	// Encode message using public key
	printf("Encrypting message...\n");
	unsigned long long ciphertext[50];
	RSA_encode(secret_message, sizeof secret_message, ciphertext, sizeof ciphertext, my_key.e, my_key.n);
	
	// Print the ciphertext
	printf("Ciphertext : ");
	for (int i = 0; i < sizeof(secret_message); i++)
	{
		if (i % 10 == 0) { printf("\n"); }
		printf("%6d ", ciphertext[i]);
	}

	// Decrypt message using private key
	printf("\n\nDecrypting using private key...\n");
	char decrypt_message[50];
	RSA_decode(ciphertext, sizeof ciphertext, decrypt_message, sizeof decrypt_message, my_key.d, my_key.n);
	printf("Decrypted message: %s\n\n", decrypt_message);

	// Attempt to bruteforce find the private key
	findPrime <<< numBlocks, numThreads >>> (my_key.n, log2(my_key.n));
	hipDeviceSynchronize();

	// Error checking
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));

	//printf("%f\n", 31243 % 10239);


	// Decrypt message using cracked key
	
	
	
	printf("\nEnd Program\n");
}

RSA_KEY generate_RSA_key(unsigned long p, unsigned long q)
{
	RSA_KEY ret_str;

	//ret_str.p = 157; // TODO: hardcoded for now - needs random generation
	//ret_str.q = 199; // TODO: hardcoded for now - needs random generation
	ret_str.p = p;
	ret_str.q = q;

	// Calculate modulus
	ret_str.n = ret_str.p * ret_str.q;

	// Calculate totient
	int totient = (ret_str.p - 1) * (ret_str.q - 1);
	printf("Totient: %d\n", totient);

	// Calculate public key exponent 'e'
	int temp_e = 0;
	while (true)
	{
		temp_e = rand() % totient + 1; // random int  1 < e < totient
		if (gcd(temp_e, totient) == 1)
		{
			break;
		}
	}
	ret_str.e = temp_e;

	// Calculate private key exponent 'd'
	int temp_d = 0;
	int diff;
	while (true)
	{
		temp_d++;
		diff = (temp_d * ret_str.e) - 1;
		if(diff % totient == 0)
		{
			break;
		}
	}
	ret_str.d = temp_d;

	return ret_str;
}

void print_RSA_key(RSA_KEY in_key)
{
	printf("RSA Key: p = %d\n", in_key.p);
	printf("RSA Key: q = %d\n", in_key.q);
	printf("RSA Key: n = %d\n", in_key.n);
	printf("RSA Key: e = %d\n", in_key.e);
	printf("RSA Key: d = %d\n", in_key.d);
	printf("\n");
}

// Greatest Common Denominator function
// Courtest of: https://codereview.stackexchange.com/a/39110
int gcd(int a, int b)
{
	int x;
	while (b)
	{
		x = a % b;
		a = b;
		b = x;
	}
	return a;
}

// RSA Message encoder
void RSA_encode(
	char *input,
	size_t input_size,
	unsigned long long *output,
	size_t output_size,
	unsigned long e,
	unsigned long n)
{
	unsigned long long m,c;
	//printf("e: %d n: %d\n", e, n);

	// Convert message string to integer
	for (int i = 0; i < input_size; i++)
	{
		m = (int)input[i]; //printf("m: %d ", m);
		//p = pow(m, e); printf("p: %d\n", p);
		//c = p % n;
		c = modulo(m, e, n);
		//printf("c: %d\n", c);
		output[i] = c;
	}
}

// RSA Message decoder
void RSA_decode(
	unsigned long long *input,
	size_t input_size,
	char *output,
	size_t output_size,
	unsigned long d,
	unsigned long n)
{
	for (int i = 0; i < output_size; i++)
	{
		output[i] = modulo(input[i], d, n);
	}
}

// Modulo Function for massive powers
// Courtest of: https://stackoverflow.com/a/36398956
int modulo(int a, int b, int n) {
	long long x = 1, y = a;
	while (b > 0) {
		if (b % 2 == 1) {
			x = (x*y) % n;
		}
		y = (y*y) % n; // squaring the base
		b /= 2;
	}
	return x%n;
}

// Test if a number is prime
__device__ int is_prime(unsigned long input)
{
	//if (input == 1)
		//return 0;

	for (unsigned long k = 2; k < input; k++)
	{
		if (input % k == 0)
		{
			return 0;
		}
			
	}
	return 1;
}
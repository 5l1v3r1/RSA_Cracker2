#include "hip/hip_runtime.h"

#include <string>
#include <math.h>

#include <stdio.h>

struct RSA_KEY
{
	unsigned long p; // selected prime 1
	unsigned long q; // selected prime 2
	unsigned long n; // public - the modulus
	unsigned long e; // public - for encryption
	unsigned long d; // private - for decryption
};

// Function prototypes
RSA_KEY generate_RSA_key();
void print_RSA_key(RSA_KEY in_key);
void RSA_encode(
	char *input,
	size_t input_size,
	unsigned long long *output,
	size_t output_size,
	unsigned long e,
	unsigned long n);
void RSA_decode(
	unsigned long long *input,
	size_t input_size,
	char *output,
	size_t output_size,
	unsigned long d,
	unsigned long n);
int gcd(int a, int b);
int modulo(int a, int b, int n);

int main()
{
	// Message to encode
	char secret_message[] = "The quick brown fox jumped over the lazy dog.";
	printf("Message: %s\n\n",secret_message);

	// Generate public & private key
	printf("Generating key...\n");
	RSA_KEY my_key;
	my_key = generate_RSA_key();
	print_RSA_key(my_key);

	// Encode message using public key
	printf("Encrypting message...\n");
	unsigned long long ciphertext[50];
	RSA_encode(secret_message, sizeof secret_message, ciphertext, sizeof ciphertext, my_key.e, my_key.n);
	
	printf("Ciphertext : ");
	for (int i = 0; i < sizeof(secret_message); i++)
	{
		if (i % 10 == 0) { printf("\n"); }
		printf("%6d ", ciphertext[i]);
	}

	// Decrypt message using private key
	printf("\n\nDecrypting using private key...\n");
	char decrypt_message[50];
	RSA_decode(ciphertext, sizeof ciphertext, decrypt_message, sizeof decrypt_message, my_key.d, my_key.n);
	printf("Decrypted message: %s\n", decrypt_message);

	// Attempt to bruteforce find the private key

	// Decrypt message using 
	
	
	
	printf("\nEnd Program\n");
}

RSA_KEY generate_RSA_key()
{
	RSA_KEY ret_str;

	ret_str.p = 11; // TODO: hardcoded for now - needs random generation
	ret_str.q = 13; // TODO: hardcoded for now - needs random generation

	// Calculate modulus
	ret_str.n = ret_str.p * ret_str.q;

	// Calculate totient
	int totient = (ret_str.p - 1) * (ret_str.q - 1);
	printf("Totient: %d\n", totient);

	// Calculate public key exponent 'e'
	int temp_e = 0;
	while (true)
	{
		temp_e = rand() % totient + 1; // random int  1 < e < totient
		if (gcd(temp_e, totient) == 1)
		{
			break;
		}
	}
	ret_str.e = temp_e;

	// Calculate private key exponent 'd'
	int temp_d = 0;
	int diff;
	while (true)
	{
		temp_d++;
		diff = (temp_d * ret_str.e) - 1;
		if(diff % totient == 0)
		{
			break;
		}
	}
	ret_str.d = temp_d;

	return ret_str;
}

void print_RSA_key(RSA_KEY in_key)
{
	printf("RSA Key: p = %d\n", in_key.p);
	printf("RSA Key: q = %d\n", in_key.q);
	printf("RSA Key: n = %d\n", in_key.n);
	printf("RSA Key: e = %d\n", in_key.e);
	printf("RSA Key: d = %d\n", in_key.d);
	printf("\n");
}

// Greatest Common Denominator function
// Courtest of: https://codereview.stackexchange.com/a/39110
int gcd(int a, int b)
{
	int x;
	while (b)
	{
		x = a % b;
		a = b;
		b = x;
	}
	return a;
}

// RSA Message encoder
void RSA_encode(
	char *input,
	size_t input_size,
	unsigned long long *output,
	size_t output_size,
	unsigned long e,
	unsigned long n)
{
	unsigned long long m,c,p;
	//printf("e: %d n: %d\n", e, n);

	// Convert message string to integer
	for (int i = 0; i < input_size; i++)
	{
		m = (int)input[i]; //printf("m: %d ", m);
		//p = pow(m, e); printf("p: %d\n", p);
		//c = p % n;
		c = modulo(m, e, n);
		//printf("c: %d\n", c);
		output[i] = c;
	}
}

// RSA Message decoder
void RSA_decode(
	unsigned long long *input,
	size_t input_size,
	char *output,
	size_t output_size,
	unsigned long d,
	unsigned long n)
{
	for (int i = 0; i < output_size; i++)
	{
		output[i] = modulo(input[i], d, n);
	}
}

// Modulo Function for massive powers
// Courtest of: https://stackoverflow.com/a/36398956
int modulo(int a, int b, int n) {
	long long x = 1, y = a;
	while (b > 0) {
		if (b % 2 == 1) {
			x = (x*y) % n;
		}
		y = (y*y) % n; // squaring the base
		b /= 2;
	}
	return x%n;
}